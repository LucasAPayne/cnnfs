#include "hip/hip_runtime.h"
#include "cuda_base.h"
#include "cnnfs_math_cuda.h"
#include "vector_cuda.h"

__global__ internal void linspace_kernel(vec<f32> v, f32 x1, f32 dx)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i >= v.elements) return;
    
    v.data[i] = x1 + ((f32)i*dx);
}

__global__ internal void linspace_kernel(vec<f64> v, f64 x1, f64 dx)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i >= v.elements) return;
    
    v.data[i] = x1 + ((f64)i*dx);
}

__global__ internal void sin_vec_kernel(vec<f32> v)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i >= v.elements) return;
    
    v.data[i] = sinf(v.data[i]);
}

__global__ internal void sin_vec_kernel(vec<f64> v)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i >= v.elements) return;
    
    v.data[i] = sin(v.data[i]);
}

__global__ internal void cos_vec_kernel(vec<f32> v)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i >= v.elements) return;
    
    v.data[i] = cosf(v.data[i]);
}

__global__ internal void cos_vec_kernel(vec<f64> v)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i >= v.elements) return;
    
    v.data[i] = cos(v.data[i]);
}

__global__ internal void exp_vec_kernel(vec<f32> v)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i >= v.elements) return;
    
    v.data[i] = expf(v.data[i]);
}

__global__ internal void exp_vec_kernel(vec<f64> v)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i >= v.elements) return;
    
    v.data[i] = exp(v.data[i]);
}

__global__ internal void exp_mat_kernel(mat<f32> m)
{
    int row = threadIdx.y + blockIdx.y*blockDim.y;
    int col = threadIdx.x + blockIdx.x*blockDim.x;
    if (row >= m.rows || col >= m.cols) return;

    size i = m.cols*row + col;
    m.data[i] = expf(m.data[i]);
}

__global__ internal void exp_mat_kernel(mat<f64> m)
{
    int row = threadIdx.y + blockIdx.y*blockDim.y;
    int col = threadIdx.x + blockIdx.x*blockDim.x;
    if (row >= m.rows || col >= m.cols) return;

    size i = m.cols*row + col;
    m.data[i] = exp(m.data[i]);
}

vec<f32> linspace_gpu(f32 x1, f32 x2, size n)
{
    vec<f32> result = vec_zeros_gpu<f32>(n);
    f32 dx = (x2 - x1) / (n - 1.0f);
    linspace_kernel<<<1, 256>>>(result, x1, dx);

    return result;
}

vec<f64> linspace_gpu(f64 x1, f64 x2, size n)
{
    vec<f64> result = vec_zeros_gpu<f64>(n);
    f64 dx = (x2 - x1) / (n - 1.0f);
    linspace_kernel<<<1, 256>>>(result, x1, dx);

    return result;
}

void sin_vec_gpu(vec<f32> v)
{
    ThreadLayout layout = calc_thread_dim(v.elements);
    sin_vec_kernel<<<layout.grid_dim, layout.block_dim>>>(v);
}

void sin_vec_gpu(vec<f64> v)
{
    ThreadLayout layout = calc_thread_dim(v.elements);
    sin_vec_kernel<<<layout.grid_dim, layout.block_dim>>>(v);
}

void cos_vec_gpu(vec<f32> v)
{
    ThreadLayout layout = calc_thread_dim(v.elements);
    cos_vec_kernel<<<layout.grid_dim, layout.block_dim>>>(v);
}

void cos_vec_gpu(vec<f64> v)
{
    ThreadLayout layout = calc_thread_dim(v.elements);
    cos_vec_kernel<<<layout.grid_dim, layout.block_dim>>>(v);
}

void exp_vec_gpu(vec<f32> v)
{
    ThreadLayout layout = calc_thread_dim(v.elements);
    exp_vec_kernel<<<layout.grid_dim, layout.block_dim>>>(v);
}

void exp_vec_gpu(vec<f64> v)
{
    ThreadLayout layout = calc_thread_dim(v.elements);
    exp_vec_kernel<<<layout.grid_dim, layout.block_dim>>>(v);
}

void exp_mat_gpu(mat<f32> m)
{
    ThreadLayout layout = calc_thread_dim(m.rows, m.cols);
    exp_mat_kernel<<<layout.grid_dim, layout.block_dim>>>(m);
}

void exp_mat_gpu(mat<f64> m)
{
    ThreadLayout layout = calc_thread_dim(m.rows, m.cols);
    exp_mat_kernel<<<layout.grid_dim, layout.block_dim>>>(m);
}
